#include <stdio.h>
#include <cassert>
#include <random>

#include "kernels.cuh"

void cuda_check_error()
{
  hipDeviceSynchronize();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("CUDA error: %s\n", hipGetErrorString(err));
  }
}

int main() {
	
	constexpr int total_size = 1 << 28;
	
	int work_per_block = total_size / (BLOCKS_PER_GRID);
	
	assert(work_per_block % (32 * 4) == 0);
	
	printf("Data size in GB: %f\n", total_size * sizeof(int) / 1e9);
	printf("Work per block: %d\n", work_per_block);
	
	int *data = new int[total_size];
	
	for (int i = 0; i < total_size; i++) {
		data[i] = rand() % 10;
	}
	
	int *d_data, *d_results;
	hipMalloc(&d_data, total_size * sizeof(int));
	
	hipMemcpy(d_data, data, total_size * sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc(&d_results, WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32 * sizeof(int));
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	for (int i = 0; i < 1000; i++) {
		#if KERNEL == 1
		kernel_stride3<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_data, total_size, work_per_block, d_results);
		#elif KERNEL == 2
		kernel_contiguous<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_data, total_size, work_per_block, d_results);
		#endif
	}
	
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	printf("Time: %f ms\n", milliseconds);
	
	cuda_check_error();
	
	int *results = new int[WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32];
	
	hipMemcpy(results, d_results, WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32 * sizeof(int), hipMemcpyDeviceToHost);
	
	int sum = 0;
	for (int i = 0; i < WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32; i++) {
		sum += results[i];
	}
	
	int cpu_sum = 0;
	for (int i = 0; i < total_size; i++) {
		cpu_sum += data[i];
	}
	
	printf("Sum: %d\n", sum);
	printf("CPU Sum: %d\n", cpu_sum);
	
	return 0;
}