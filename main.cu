#include <stdio.h>
#include <cassert>

#include "kernels.cuh"


int main() {
	
	constexpr int total_size = 1 << 28;
	
	int work_per_warp = total_size / (WARPS_PER_BLOCK * BLOCKS_PER_GRID);
	
	assert(work_per_warp % (32 * 4) == 0);
	
	printf("Data size in GB: %f\n", total_size * sizeof(int) / 1e9);
	printf("Work per warp: %d\n", work_per_warp);
	
	int *data = new int[total_size];
	
	for (int i = 0; i < total_size; i++) {
		data[i] = i % 5 + 1;
	}
	
	int *d_data, *d_results;
	hipMalloc(&d_data, total_size * sizeof(int));
	
	hipMemcpy(d_data, data, total_size * sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc(&d_results, WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32 * sizeof(int));
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	for (int i = 0; i < 1000; i++) {
		kernel_stride<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_data, total_size, work_per_warp, d_results);
	}
	
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	printf("Time: %f ms\n", milliseconds);
	
	int *results = new int[WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32];
	
	hipMemcpy(results, d_results, WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32 * sizeof(int), hipMemcpyDeviceToHost);
	
	int sum = 0;
	for (int i = 0; i < WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32; i++) {
		sum += results[i];
	}
	
	int cpu_sum = 0;
	for (int i = 0; i < total_size; i++) {
		cpu_sum += data[i];
	}
	
	printf("Sum: %d\n", sum);
	printf("CPU Sum: %d\n", cpu_sum);
	
	return 0;
}