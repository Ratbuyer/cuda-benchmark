#include <stdio.h>
#include <cassert>
#include <random>
#include <algorithm>

#include "kernels.cuh"

void cuda_check_error()
{
  hipDeviceSynchronize();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("CUDA error: %s\n", hipGetErrorString(err));
  }
}

int main() {
	#if KERNEL == 1
	constexpr int slot_size = 32 * 4 * BLOCK_SIZE;
	#elif KERNEL == 2
	constexpr int slot_size = 32 * BLOCK_SIZE;
	#endif
	
	constexpr int total_size = (1 << DATA_SIZE) * WARPS_PER_BLOCK * BLOCKS_PER_GRID;
	
	int num_slots = total_size / slot_size;
	int slots_per_warp = num_slots / (WARPS_PER_BLOCK * BLOCKS_PER_GRID);
	
	printf("Data size in GB: %f\n", total_size * sizeof(int) / 1e9);
	printf("num slots: %d\n", num_slots);
	printf("slots per warp: %d\n", slots_per_warp);
	
	assert(total_size % slot_size == 0);
	assert(num_slots % (WARPS_PER_BLOCK * BLOCKS_PER_GRID) == 0);
	
	int *slots = new int[num_slots];
	
	for (int i = 0; i < num_slots; i++) {
		slots[i] = i;
	}
	
	int *data = new int[total_size];
	
	for (int i = 0; i < total_size; i++) {
		data[i] = rand() % 10;
	}
	
	// permutate the num_slots
	std::random_shuffle(slots, slots + num_slots);
	
	int *d_data, *d_results;
	hipMalloc(&d_data, total_size * sizeof(int));
	hipMemcpy(d_data, data, total_size * sizeof(int), hipMemcpyHostToDevice);
	
	int *d_slots;
	hipMalloc(&d_slots, num_slots * sizeof(int));
	hipMemcpy(d_slots, slots, num_slots * sizeof(int), hipMemcpyHostToDevice);	
	
	hipMalloc(&d_results, WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32 * sizeof(int));
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	for (int i = 0; i < 1000; i++) {
		#if KERNEL == 1
		kernel_int4<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_data, d_slots, slots_per_warp, num_slots, d_results);
		#elif KERNEL == 2
		kernel_int<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_data, d_slots, slots_per_warp, num_slots, d_results);
		#endif
	}
	
	cuda_check_error();
	
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	printf("Time: %f ms\n", milliseconds);
	
	int *results = new int[WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32];
	
	hipMemcpy(results, d_results, WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32 * sizeof(int), hipMemcpyDeviceToHost);
	
	int sum = 0;
	for (int i = 0; i < WARPS_PER_BLOCK * BLOCKS_PER_GRID * 32; i++) {
		sum += results[i];
	}
	
	int cpu_sum = 0;
	for (int i = 0; i < total_size; i++) {
		cpu_sum += data[i];
	}
	
	printf("Sum: %d\n", sum);
	printf("CPU Sum: %d\n", cpu_sum);
	
	assert(sum == cpu_sum);
	
	return 0;
}